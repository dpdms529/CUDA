

#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 512

__global__ void device_add(int *a, int *b, int *c){
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void fill_array(int *data){
    for(int idx=0;idx<N;idx++){
        data[idx] = idx;
    }
}

void print_output(int *a, int *b, int *c){
    for(int idx = 0; idx<N; idx++){
        printf("\n %d + %d = %d", a[idx], b[idx], c[idx]);
    }
}

int main(void){
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;

    int size = N * sizeof(int);

    a = (int *)malloc(size); fill_array(a);
    b = (int *)malloc(size); fill_array(b);
    c = (int *)malloc(size);

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    device_add<<<N,1>>>(d_a,d_b,d_c);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    print_output(a,b,c);
    
    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c); 

    return 0;
}